#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

__global__ void prefix_sum(int * arr, int N)
{
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  for(int off = 1; off < N ; off *= 2)
  {
    if(idx >= off)
    {
      int val = idx % (2 * off);

      if(val >= off)
      {
        arr[idx] += arr[idx - val + off -1];
      }
    }
  }
}

int main(void)
{
    int facs[5] = {2, 3, 5, 6 , 9};
    int *dfacs;

    hipMalloc(&dfacs, 5 * sizeof(int));
    hipMemcpy(dfacs, facs, 5 * sizeof(int), hipMemcpyHostToDevice);

    prefix_sum<<<1,5>>>(dfacs, 5);

    hipMemcpy(facs, dfacs, 5 * sizeof(int), hipMemcpyDeviceToHost);

    for(int i =0; i< 5; i++)
    {
        printf("%d\n", facs[i]);
    }
}